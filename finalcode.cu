#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <hiprand/hiprand_kernel.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include <thrust/device_ptr.h>
#include <sys/time.h>
using namespace std;

__global__ void partition_step (hiprandState * state, unsigned long seed )
{
    int i= blockDim.x * blockIdx.x + threadIdx.x;
    hiprand_init (seed, i, 0, &state[i]);
} 

__global__ void randomColouring (hiprandState* globalState, int *degreeCount, int n, int limit){

	int i= blockDim.x * blockIdx.x + threadIdx.x;
	
	hiprandState localState = globalState[i];
    	float RANDOM = hiprand_uniform( &localState );
    	globalState[i] = localState;
    	
    	RANDOM *= (limit - 1 + 0.999999);
    	RANDOM += 1;
	
	degreeCount[i] = (int) RANDOM;
}

__global__ void conflictDetection (int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m, int *detectConflict){

	int i= blockDim.x * blockIdx.x + threadIdx.x;
	
	if (i>=n){
		return;
	}
	
	int myColour = degreeCount[i];
	
	int incoming = -1, stop = -1;
	
	incoming = vertexArray[i];
	
	if (i==n-1){	
		stop = m;
	}
	
	else{
		stop = vertexArray[i+1];
	}
	
	for (int j=incoming; j<stop; j++){
		if (degreeCount[neighbourArray[j]-1] == myColour){

			detectConflict[i]=1;
			break;
		}
	}
}


__global__ void degreeCalc (int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m){

	int i= blockDim.x * blockIdx.x + threadIdx.x;
	
	if (i>=n){
		return;
	}
	
	
	int incoming = -1, stop = -1;
	int diff=0;
	
	incoming = vertexArray[i];
	
	if (i==n-1){	
		stop = m;
	}
	
	else{
		stop = vertexArray[i+1];
	}

	diff = stop-incoming;
		
	atomicAdd(&degreeCount[i], diff);
	
	for (int j=incoming; j<stop; j++){
		atomicAdd(&degreeCount[neighbourArray[j]-1], 1);
	}
}

int main(int argc, char const *argv[])
{

	int n, m;
	// Enter number of vertices and edges
	cin>>n>>m;

	int h_vertexArray[n];
	int h_neighbourArray[m];
	int h_degreeCount[n];
	int h_detectConflict[n];
	
	
	// Cuda memory allocation
	size_t bytes = n*sizeof(int);
    int *d_vertexArray = NULL;
    hipMalloc((void **)&d_vertexArray, bytes);
    
    int *d_neighbourArray = NULL;
    hipMalloc((void **)&d_neighbourArray, m*sizeof(int));
    
    int *d_detectConflict = NULL;
    hipMalloc((void **)&d_detectConflict, bytes);
    hipMemset((void *)d_detectConflict, 0,bytes);
    
    int *d_degreeCount = NULL;
    hipMalloc((void **)&d_degreeCount, bytes);
    hipMemset((void *)d_degreeCount, 0, bytes);
    
    hiprandState* partition_states;
    hipMalloc ( &partition_states, n*sizeof( hiprandState ) );
    	
	for (int i = 0; i < n; ++i)
	{
		/* code */
		h_vertexArray[i]=m;
	}

	int temp = 0;

	int current = 0;
	int mark = 1;
// Add the graph based on input file
	for (int i = 0; i < m; ++i)
	{
		/* code */

		int incoming;
		int end;

		cin>>incoming>>end;
		incoming++;
		end++;

		if (incoming!=mark){ 

			if (incoming == mark+1 && h_vertexArray[mark-1]!=m){ 

			}

			else{

				for (int j = mark; j<incoming; j++){ 
					h_vertexArray[j-1]=temp;
					
				}
			}
			mark = incoming;

		}

		if (incoming==current){ 
			h_neighbourArray[temp]=end;
			temp++;
		}

		else { 
			current = incoming;

			h_vertexArray[current-1]=temp;

			h_neighbourArray[temp]=end;
			temp++;
		}
	}

	
	hipMemcpy(d_vertexArray, h_vertexArray, n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_neighbourArray, h_neighbourArray, m*sizeof(int), hipMemcpyHostToDevice);
	
	int threadsPerBlock = 512;
	int blocksPerGrid = (n + threadsPerBlock -1)/threadsPerBlock;
	struct timeval startTime;
	struct timeval endTime;
	struct timezone startZone;
	struct timezone endZone;
	long startt,endt;
	double overhead;
	cout<<threadsPerBlock<<" "<<blocksPerGrid<<endl;
	gettimeofday(&startTime,&startZone);
	// Step 0 : Calculate degree of each vertex
	degreeCalc<<<blocksPerGrid, threadsPerBlock>>>(d_vertexArray, d_neighbourArray, d_degreeCount, n, m);

	thrust::device_ptr<int> d_ptr = thrust::device_pointer_cast(d_degreeCount);
  	int max = *(thrust::max_element(d_ptr, d_ptr + n));
	
	cout<<"Max number of colours = "<<max<<endl;


	partition_step <<<blocksPerGrid, threadsPerBlock>>> ( partition_states, time(NULL) );
	
	// Step 1 - Randomly assign colours
	randomColouring<<<blocksPerGrid, threadsPerBlock>>>(partition_states, d_degreeCount, n, max);

	hipMemcpy(h_degreeCount, d_degreeCount, n*sizeof(int), hipMemcpyDeviceToHost);
    cout<<"randomColouring"<<endl;
	for (int i=0; i<n; i++){
		cout<<"Color of"<<i<<": "<<h_degreeCount[i]<<endl;
	}
	cout<<endl;
	conflictDetection<<<blocksPerGrid, threadsPerBlock>>>(d_vertexArray, d_neighbourArray, d_degreeCount, n, m, d_detectConflict);
	
	thrust::device_ptr<int> d_detectConflict_ptr = thrust::device_pointer_cast(d_detectConflict);
  	int count1 = thrust::reduce(d_detectConflict_ptr, d_detectConflict_ptr + n);
  	
  	hipMemcpy(h_detectConflict, d_detectConflict, n*sizeof(int), hipMemcpyDeviceToHost);
	
	int countnew=0;
	int old_colors[n];
	for (int i = 0; i < n; ++i)
	{
		/* code */
		old_colors[i] = -1;
	}
	for (int i=0; i<n-1; i++){
		
		if (h_detectConflict[i]==0){
			continue;
		}
		
		countnew++;
		
		bool usedColours[n];
		
		fill(usedColours, usedColours+n, false);
		

		
		int incoming = -1, stop = -1;
	
		incoming = h_vertexArray[i];
		
		stop = h_vertexArray[i+1];
		old_colors[i] = h_degreeCount[i];
		
		
		for (int j=incoming; j<stop; j++){
		

			usedColours[h_degreeCount[h_neighbourArray[j]-1]-1] = true;
		}

		
		for (int j=0; j<n; j++){
			if (usedColours[j]==false){
				h_degreeCount[i]=j+1;
				break;
			}
		}
	}
	
	
	
	if (h_detectConflict[n-1]!=0){

		bool usedColours[n];
		
		countnew++;
		
		fill(usedColours, usedColours+n, false);
		
		int incoming = -1, stop = -1;
	
		incoming = h_vertexArray[n-1];
	
		stop = m;
		
	
		for (int j=incoming; j<stop; j++){
			usedColours[h_degreeCount[h_neighbourArray[j]-1]-1] = true;
		}
		
		for (int j=0; j<n; j++){
			if (usedColours[j]==false){
				h_degreeCount[n-1]=j+1;
				break;
			}
		}
	}
	for (int i = 0; i < n; ++i)
	{
		cout<<"Colour of i from" <<i <<" "<<old_colors[i]<<":"<<h_degreeCount[i]<<endl;
	}
	

	hipMemset((void *)d_detectConflict, 0, (n)*sizeof(int));
	
	hipMemcpy(d_degreeCount, h_degreeCount, n*sizeof(int), hipMemcpyHostToDevice);



	conflictDetection<<<blocksPerGrid, threadsPerBlock>>>(d_vertexArray, d_neighbourArray, d_degreeCount, n, m, d_detectConflict);
	gettimeofday(&endTime,&endZone);
	startt = startTime.tv_sec*1000000+startTime.tv_usec;
	endt = endTime.tv_sec*1000000+endTime.tv_usec;
	overhead = (endt-startt)/1000000.0;
	count1 = thrust::reduce(d_detectConflict_ptr, d_detectConflict_ptr + n);
  		
	cout<<"Count: "<<count1<<"    "<<countnew<<endl;
	cout<<"time taken is"<<overhead<<endl;

	hipFree(d_neighbourArray);
	hipFree(d_vertexArray);
	hipFree(d_degreeCount);
	hipFree(d_detectConflict);
	
	hipDeviceReset();
	return 0; 

}
